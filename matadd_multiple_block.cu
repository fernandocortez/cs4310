#include "hip/hip_runtime.h"
/*
 * The purpose of this program is to add two matrices with multiple thread
 * blocks on a GPU written in CUDA C.
 */

#include <stdio.h>
#include "util.h"

__global__ void matrixAdd(float *A, float *B, float *C, size_t size, size_t width)
{
    size_t bx = blockIdx.x;
    size_t by = blockIdx.y;
    size_t tx = threadIdx.x;
    size_t ty = threadIdx.y;

    size_t row = by * width + ty;
    size_t column = bx * width + tx;
    size_t coordinate = row + column;
    int temp;

    temp = A[coordinate] + B[coordinate];
    C[coordinate] = temp;
}

int main(int argc, char *argv[])
{
    size_t i, j;
    size_t width;
    size_t size, total_size;
    int memory_size;

    float *matrixA = NULL;
    float *matrixB = NULL;
    float *matrixC = NULL;
    float *data = NULL;

    if(argc != 3) {
        printf("format:%s [size of matrix] [size of small matrix]\n", argv[0]);
        exit(1);
    }

    size = (unsigned) atoi(argv[1]);
    width = (unsigned) atoi(argv[2]);
    total_size = size * size;
    memory_size = total_size * sizeof(float);

    /* allocate host memory */
    data = (float*) malloc(memory_size);

    /* allocate device memory */
    (hipMalloc( (void**) &matrixA, memory_size));
    (hipMalloc( (void**) &matrixB, memory_size));
    (hipMalloc( (void**) &matrixC, memory_size));
    checkErrors("Memory allocation\n");

    for(i = 0; i < total_size; i++)
        data[i] = 1.0; /* (int) (10 * rand()/32768.f); */

    if(size < 6) {
        for(i = 0; i < size; i++) {
            for(j = 0; j < size; j++)
                printf("%3.2f", data[i*size + j]);
            printf("\n");
        }
    }

    /* copy data from host memory to device memory */
    (hipMemcpy( matrixA, data, memory_size, hipMemcpyHostToDevice ));
    (hipMemcpy( matrixB, data, memory_size, hipMemcpyHostToDevice ));
    checkErrors("Memory copy 1\n");

    dim3 dimBlock(width, width);
    dim3 dimGrid(size/width, size/width);

    /* timing */
    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    hipEventRecord(start_event, 0);

    /* call kernel (global function) */
    matrixAdd<<<dimGrid, dimBlock>>>(matrixA, matrixB, matrixC, size, width);
    hipDeviceSynchronize();

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    float time_kernel;
    hipEventElapsedTime(&time_kernel, start_event, stop_event);
    printf("Total time %f\n", time_kernel);

    /* copy data from device memory to host memory */
    (hipMemcpy( data, matrixC, memory_size, hipMemcpyDeviceToHost ));
    checkErrors("Memory copy 2\n");

    if(size < 6) {
        for(i = 0; i < size; i++) {
            for(j = 0; j < size; j++)
                printf("%d", data[i*size + j]);
            printf("\n");
        }
    }

    free(data);
    data = NULL;
    hipFree(matrixA);
    matrixA = NULL;
    hipFree(matrixB);
    matrixB = NULL;
    hipFree(matrixC);
    matrixC = NULL;

    exit(0);
}
